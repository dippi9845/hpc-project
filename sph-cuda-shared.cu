#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * sph.c -- Smoothed Particle Hydrodynamics
 *
 * https://github.com/cerrno/mueller-sph
 *
 * Copyright (C) 2016 Lucas V. Schuermann
 * Copyright (C) 2022 Moreno Marzolla
 *
 * Permission is hereby granted, free of charge, to any person
 * obtaining a copy of this software and associated documentation
 * files (the "Software"), to deal in the Software without
 * restriction, including without limitation the rights to use, copy,
 * modify, merge, publish, distribute, sublicense, and/or sell copies
 * of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS
 * BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN
 * ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

/* "Particle-Based Fluid Simulation for Interactive Applications" by
   Müller et al. solver parameters */

const float Gx = 0.0, Gy = -10.0;   // external (gravitational) forces
const float REST_DENS = 300;    // rest density
const float GAS_CONST = 2000;   // const for equation of state
const float H = 16;             // kernel radius
const float EPS = 16;           // equal to H
const float MASS = 2.5;         // assume all particles have the same mass
const float VISC = 200;         // viscosity constant
const float DT = 0.0007;        // integration timestep
const float BOUND_DAMPING = -0.5;

// rendering projection parameters
// (the following ought to be "const float", but then the compiler
// would give an error because VIEW_WIDTH and VIEW_HEIGHT are
// initialized with non-literal expressions)

const int MAX_PARTICLES = 20000;
// Larger window size to accommodate more particles
#define WINDOW_WIDTH 3000
#define WINDOW_HEIGHT 2000

const int DAM_PARTICLES = 500;

const float VIEW_WIDTH = 1.5 * WINDOW_WIDTH;
const float VIEW_HEIGHT = 1.5 * WINDOW_HEIGHT;

#define PRINT_AVERANGE 10

#define BLKDIM 1024

/* Particle data structure; stores position, velocity, and force for
   integration stores density (rho) and pressure values for SPH.

   You may choose a different layout of the particles[] data structure
   to suit your needs. */
typedef struct {
    float x, y;         // position
    float vx, vy;       // velocity
    float fx, fy;       // force
    float rho, p;       // density, pressure
} particle_t;


float *pos_x, *pos_y;
float *vx, *vy;
float *fx, *fy;
float *rho, *p; 

int n_particles = 0;    // number of currently active particles

#define SHARED_MEM_PER_BLOCK 49152

/**
 * Return a random value in [a, b]
 */
float randab(float a, float b)
{
    return a + (b-a)*rand() / (float)(RAND_MAX);
}

/**
 * Set initial position of particle `*p` to (x, y); initialize all
 * other attributes to default values (zeros).
 */
void init_particle( int index , float x, float y )
{
    pos_x[index] = x;
    pos_y[index] = y;
    vx[index] = vy[index] = 0.0;
    fx[index] = fy[index] = 0.0;
    rho[index] = 0.0;
    p[index] = 0.0;
}

/**
 * Return nonzero iff (x, y) is within the frame
 */
int is_in_domain( float x, float y )
{
    return ((x < VIEW_WIDTH - EPS) &&
            (x > EPS) &&
            (y < VIEW_HEIGHT - EPS) &&
            (y > EPS));
}

/**
 * Initialize the SPH model with `n` particles. The caller is
 * responsible for allocating the `particles[]` array of size
 * `MAX_PARTICLES`.
 *
 * DO NOT parallelize this function, since it calls rand() which is
 * not thread-safe.
 *
 * For MPI and OpenMP: only the master must initialize the domain;
 *
 * For CUDA: the CPU must initialize the domain.
 */
void init_sph( int n )
{
    n_particles = 0;
    //printf("Initializing with %d particles\n", n);

    for (float y = EPS; y < VIEW_HEIGHT - EPS; y += H) {
        for (float x = EPS; x <= VIEW_WIDTH * 0.8f; x += H) {
            if (n_particles < n) {
                float jitter = rand() / (float)RAND_MAX;
                init_particle(n_particles, x+jitter, y);
                n_particles++;
            } else {
                return;
            }
        }
    }
    assert(n_particles == n);
}

/**
 ** You may parallelize the following four functions
 **/

__global__ void compute_density_pressure( float* d_rho, float* d_pos_x, float * d_pos_y, float * d_p, int n_particles)
{
    const int index_particle = threadIdx.x + blockIdx.x * blockDim.x;
    const int lindex = threadIdx.x;
    const int FLOAT_PER_SHARED_MEM = SHARED_MEM_PER_BLOCK / sizeof(float);
    
    
    __shared__ float sh_pos_x[FLOAT_PER_SHARED_MEM/2];
    __shared__ float sh_pos_y[FLOAT_PER_SHARED_MEM/2];

    const float HSQ = H * H;    // radius^2 for optimization

    /* Smoothing kernels defined in Muller and their gradients adapted
    to 2D per "SPH Based Shallow Water Simulation" by Solenthaler
    et al. */
    const float POLY6 = 4.0 / (M_PI * pow(H, 8));

    d_rho[index_particle] = 0.0;
    
    // per ogni particella memorizzi 2 float
    // numero di volte di cui devi fare una copia nella shared per tutto il kernel
    const int repetitions = (n_particles * 2 + FLOAT_PER_SHARED_MEM - 1) / FLOAT_PER_SHARED_MEM;
    const int max_particles_to_copy = FLOAT_PER_SHARED_MEM / 2;

    printf("[idx: %4d] [x: %f] [y: %f] [rho: %f] [p: %f]\n",
            index_particle,
            d_pos_x[index_particle],
            d_pos_y[index_particle],
            d_rho[index_particle],
            d_p[index_particle]
            );
    
    for (int r = 0; r < repetitions;  r++) {
        int end_copy = max_particles_to_copy;

        if (r == repetitions - 1) {
            end_copy = n_particles - max_particles_to_copy * r;
        }

        int copy_shift = 0;
        while (copy_shift * BLKDIM + lindex < end_copy) {
            sh_pos_x[copy_shift * BLKDIM + lindex] = d_pos_x[r * max_particles_to_copy + copy_shift * BLKDIM + lindex];
            sh_pos_y[copy_shift * BLKDIM + lindex] = d_pos_y[r * max_particles_to_copy + copy_shift * BLKDIM + lindex];
            copy_shift++;
        }

        __syncthreads();

        if (index_particle < n_particles) {
            for (int j = 0; j < end_copy; j++) {

                const float dx = sh_pos_x[j] - d_pos_x[index_particle];
                const float dy = sh_pos_y[j] - d_pos_y[index_particle];
                const float d2 = dx*dx + dy*dy;

                if (d2 < HSQ) {
                    d_rho[index_particle] += MASS * POLY6 * pow(HSQ - d2, 3.0);
                }
            }

            d_p[index_particle] = GAS_CONST * (d_rho[index_particle] - REST_DENS);
        }

        __syncthreads();
    }
}

__global__ void compute_forces( float* d_rho, float* d_pos_x, float * d_pos_y, float * d_p, float* d_vx, float* d_vy, float* d_fx, float* d_fy, int n_particles )
{
    const int index_particle = threadIdx.x + blockIdx.x * blockDim.x;
    /* Smoothing kernels defined in Muller and their gradients adapted
       to 2D per "SPH Based Shallow Water Simulation" by Solenthaler
       et al. */
    const int lindex = threadIdx.x;
    const int FLOAT_PER_SHARED_MEM = SHARED_MEM_PER_BLOCK / sizeof(float);

    const float SPIKY_GRAD = -10.0 / (M_PI * pow(H, 5));
    const float VISC_LAP = 40.0 / (M_PI * pow(H, 5));
    const float EPS = 1e-6;

    float fpress_x = 0.0, fpress_y = 0.0;
    float fvisc_x = 0.0, fvisc_y = 0.0;

    __shared__ float sh_pos_x[FLOAT_PER_SHARED_MEM/2];
    __shared__ float sh_pos_y[FLOAT_PER_SHARED_MEM/2];

    const int repetitions = (n_particles * 2 + FLOAT_PER_SHARED_MEM - 1) / FLOAT_PER_SHARED_MEM;
    const int max_particles_to_copy = FLOAT_PER_SHARED_MEM / 2;

    printf("[idx: %4d] [vx: %f] [vy: %f] [x: %f] [y: %f] [fx: %f] [fy: %f] [rho: %f] [p: %f]\n",
                index_particle,
                d_vx[index_particle],
                d_vy[index_particle],
                d_pos_x[index_particle],
                d_pos_y[index_particle],
                d_fx[index_particle],
                d_fy[index_particle],
                d_rho[index_particle],
                d_p[index_particle]
                );
    
    for (int r = 0; r < repetitions;  r++) {
        int end_copy = max_particles_to_copy;

        if (r == repetitions - 1) {
            end_copy = n_particles - max_particles_to_copy * r;
        }

        int copy_shift = 0;
        while (copy_shift * BLKDIM + lindex < end_copy) {
            sh_pos_x[copy_shift * BLKDIM + lindex] = d_pos_x[r * max_particles_to_copy + copy_shift * BLKDIM + lindex];
            sh_pos_y[copy_shift * BLKDIM + lindex] = d_pos_y[r * max_particles_to_copy + copy_shift * BLKDIM + lindex];
            copy_shift++;
        }

        __syncthreads();

        if (index_particle < n_particles)  {
            for (int j=0; j< end_copy; j++) {

                if (index_particle == r * max_particles_to_copy + j)
                    continue;

                const float dx = sh_pos_x[j] - d_pos_x[index_particle];
                const float dy = sh_pos_y[j] - d_pos_y[index_particle];
                const float dist = hypotf(dx, dy) + EPS; // avoids division by zero later on

                if (dist < H) {
                    const float norm_dx = dx / dist;
                    const float norm_dy = dy / dist;
                    // compute pressure force contribution
                    fpress_x += -norm_dx * MASS * (d_p[index_particle] + d_p[j]) / (2 * d_rho[j]) * SPIKY_GRAD * pow(H - dist, 3);
                    fpress_y += -norm_dy * MASS * (d_p[index_particle] + d_p[j]) / (2 * d_rho[j]) * SPIKY_GRAD * pow(H - dist, 3);
                    // compute viscosity force contribution
                    fvisc_x += VISC * MASS * (d_vx[j] - d_vx[index_particle]) / d_rho[j] * VISC_LAP * (H - dist);
                    fvisc_y += VISC * MASS * (d_vy[j] - d_vy[index_particle]) / d_rho[j] * VISC_LAP * (H - dist);
                }
            }
        }

        __syncthreads();


    }
    const float fgrav_x = Gx * MASS / d_rho[index_particle];
    const float fgrav_y = Gy * MASS / d_rho[index_particle];
    d_fx[index_particle] = fpress_x + fvisc_x + fgrav_x;
    d_fy[index_particle] = fpress_y + fvisc_y + fgrav_y;
    
}

__global__ void integrate( float* d_rho, float* d_x, float * d_y, float* d_vx, float* d_vy, float* d_fx, float* d_fy, int n_particles )
{
    const int index_particle = threadIdx.x + blockIdx.x * blockDim.x;
    if (index_particle < n_particles) {
        printf("[idx: %4d] [vx: %f] [vy: %f] [x: %f] [y: %f] [fx: %f] [fy: %f] [rho: %f]\n",
                index_particle,
                d_vx[index_particle],
                d_vy[index_particle],
                d_x[index_particle],
                d_y[index_particle],
                d_fx[index_particle],
                d_fy[index_particle],
                d_rho[index_particle]
                );
        // forward Euler integration
        d_vx[index_particle] += DT * d_fx[index_particle] / d_rho[index_particle];
        d_vy[index_particle] += DT * d_fy[index_particle] / d_rho[index_particle];
        d_x[index_particle] += DT * d_vx[index_particle];
        d_y[index_particle] += DT * d_vy[index_particle];

        // enforce boundary conditions
        if (d_x[index_particle] - EPS < 0.0) {
            d_vx[index_particle] *= BOUND_DAMPING;
            d_x[index_particle] = EPS;
        }
        if (d_x[index_particle] + EPS > VIEW_WIDTH) {
            d_vx[index_particle] *= BOUND_DAMPING;
            d_x[index_particle] = VIEW_WIDTH - EPS;
        }
        if (d_y[index_particle] - EPS < 0.0) {
            d_vy[index_particle] *= BOUND_DAMPING;
            d_y[index_particle] = EPS;
        }
        if (d_y[index_particle] + EPS > VIEW_HEIGHT) {
            d_vy[index_particle] *= BOUND_DAMPING;
            d_y[index_particle] = VIEW_HEIGHT - EPS;
        }
    }
}

__global__ void reduction(float* d_vx, float* d_vy, int n, float * d_sums) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;

    printf("[idx: %4d] [vx: %f] [vy: %f]\n", index, d_vx[index], d_vy[index]);

    /* reduction of averange velocity */
    if (index < n) {
        __shared__ float temp[BLKDIM];
        const int lindex = threadIdx.x;
        const int bindex = blockIdx.x;
        int bsize = blockDim.x / 2;
        temp[lindex] = hypot(d_vx[index], d_vy[index]) / n;

        __syncthreads();
        while ( bsize > 0 ) {
            if ( lindex < bsize ) {
                temp[lindex] += temp[lindex + bsize];
            }
            bsize = bsize / 2;
            __syncthreads();
        }
        if ( 0 == lindex ) {
            d_sums[bindex] = temp[0];
        }
    }
}

#define MAX_BLOCK (MAX_PARTICLES + BLKDIM - 1)/BLKDIM

int main(int argc, char **argv)
{
    srand(1234);

    int n = DAM_PARTICLES;
    int nsteps = 50;

    if (argc > 3) {
        fprintf(stderr, "Usage: %s [nparticles [nsteps]]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if (argc > 1) {
        n = atoi(argv[1]);
    }

    if (argc > 2) {
        nsteps = atoi(argv[2]);
    }

    if (n > MAX_PARTICLES) {
        fprintf(stderr, "FATAL: the maximum number of particles is %d\n", MAX_PARTICLES);
        return EXIT_FAILURE;
    }

    pos_x = (float *) malloc(n * sizeof(float)); assert( pos_x != NULL );
    pos_y = (float *) malloc(n * sizeof(float)); assert( pos_y != NULL );
    vx = (float *) malloc(n * sizeof(float)); assert( vx != NULL );
    vy = (float *) malloc(n * sizeof(float)); assert( vy != NULL );
    fx = (float *) malloc(n * sizeof(float)); assert( fx != NULL );
    fy = (float *) malloc(n * sizeof(float)); assert( fy != NULL );
    rho = (float *) malloc(n * sizeof(float)); assert( rho != NULL );
    p = (float *) malloc(n * sizeof(float)); assert( p != NULL );
    
    float *d_pos_x, *d_pos_y;
    float *d_vx, *d_vy;
    float *d_fx, *d_fy;
    float *d_rho, *d_p; 


    float h_sums[MAX_BLOCK];
    float *d_sums;

    int block_num = (n + BLKDIM - 1)/BLKDIM;

    init_sph(n);

    hipMalloc((void **) &d_pos_x, sizeof(float) * n);
    hipMemcpy(d_pos_x, pos_x, sizeof(float) * n, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_pos_y, sizeof(float) * n);
    hipMemcpy(d_pos_y, pos_y, sizeof(float) * n, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_vx, sizeof(float) * n);
    hipMemcpy(d_vx, vx, sizeof(float) * n, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_vy, sizeof(float) * n);
    hipMemcpy(d_vy, vy, sizeof(float) * n, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_fx, sizeof(float) * n);
    hipMemcpy(d_fx, fx, sizeof(float) * n, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_fy, sizeof(float) * n);
    hipMemcpy(d_fy, fy, sizeof(float) * n, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_rho, sizeof(float) * n);
    hipMemcpy(d_rho, rho, sizeof(float) * n, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_p, sizeof(float) * n);
    hipMemcpy(d_p, p, sizeof(float) * n, hipMemcpyHostToDevice);
    
    hipMalloc((void **) &d_sums, block_num * sizeof(float));

    double loop_start = hpc_gettime();
    
    for (int s=0; s<nsteps; s++) {
        double start = hpc_gettime();

        printf("Pressione e Densita:\n");

        compute_density_pressure<<<block_num, BLKDIM>>>(d_rho, d_pos_x, d_pos_y, d_p, n);
        
        hipDeviceSynchronize();

        printf("Forze:\n");

        compute_forces<<<block_num, BLKDIM>>>(d_rho, d_pos_x, d_pos_y, d_p, d_vx, d_vy, d_fx, d_fy, n);

        hipDeviceSynchronize();

        printf("Integrazione");

        integrate<<<block_num, BLKDIM>>>(d_rho, d_pos_x, d_pos_y, d_vx, d_vy, d_fx, d_fy, n);

        hipDeviceSynchronize();

        printf("Reduction");

        reduction<<<block_num, BLKDIM>>>(d_vx, d_vy, n, d_sums);
        /* the average velocities MUST be computed at each step, even
        if it is not shown (to ensure constant workload per
        iteration) */
        hipMemcpy(h_sums, d_sums, block_num * sizeof(float), hipMemcpyDeviceToHost);
        
        float avg = 0.0;
        
        for (int i = 0; i < block_num; i++)
            avg += h_sums[i];
        
        double end = hpc_gettime() - start;

        if (s % PRINT_AVERANGE == 0){
            printf("step %5d, avgV=%f, took: %fs\n", s, avg, end);
            //printf("%f;", avg);
            //for (int i = 0; i < MAX_BLOCK; i++)
            //    printf("%f ", h_sums[i]);
            //printf("\n");
        }
    }

    double loop_end = hpc_gettime() - loop_start;
    printf("took: %fs\n", loop_end);

    hipFree(d_rho);
    hipFree(d_pos_x);
    hipFree(d_pos_y);
    hipFree(d_p);
    hipFree(d_vx);
    hipFree(d_vy);
    hipFree(d_fx);
    hipFree(d_fy);

    hipFree(d_sums);


    free(rho);
    free(pos_x);
    free(pos_y);
    free(p);
    free(vx);
    free(vy);
    free(fx);
    free(fy);
    return EXIT_SUCCESS;
}
