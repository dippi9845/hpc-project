#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * sph.c -- Smoothed Particle Hydrodynamics
 *
 * https://github.com/cerrno/mueller-sph
 *
 * Copyright (C) 2016 Lucas V. Schuermann
 * Copyright (C) 2022 Moreno Marzolla
 *
 * Permission is hereby granted, free of charge, to any person
 * obtaining a copy of this software and associated documentation
 * files (the "Software"), to deal in the Software without
 * restriction, including without limitation the rights to use, copy,
 * modify, merge, publish, distribute, sublicense, and/or sell copies
 * of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS
 * BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN
 * ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

/* "Particle-Based Fluid Simulation for Interactive Applications" by
   Müller et al. solver parameters */

const float Gx = 0.0, Gy = -10.0;   // external (gravitational) forces
const float REST_DENS = 300;    // rest density
const float GAS_CONST = 2000;   // const for equation of state
const float H = 16;             // kernel radius
const float EPS = 16;           // equal to H
const float MASS = 2.5;         // assume all particles have the same mass
const float VISC = 200;         // viscosity constant
const float DT = 0.0007;        // integration timestep
const float BOUND_DAMPING = -0.5;

// rendering projection parameters
// (the following ought to be "const float", but then the compiler
// would give an error because VIEW_WIDTH and VIEW_HEIGHT are
// initialized with non-literal expressions)

const int MAX_PARTICLES = 20000;
// Larger window size to accommodate more particles
#define WINDOW_WIDTH 3000
#define WINDOW_HEIGHT 2000

const int DAM_PARTICLES = 500;

const float VIEW_WIDTH = 1.5 * WINDOW_WIDTH;
const float VIEW_HEIGHT = 1.5 * WINDOW_HEIGHT;

#define PRINT_AVERANGE 10

#define BLKDIM 1024

/* Particle data structure; stores position, velocity, and force for
   integration stores density (rho) and pressure values for SPH.

   You may choose a different layout of the particles[] data structure
   to suit your needs. */
typedef struct {
    float x, y;         // position
    float vx, vy;       // velocity
    float fx, fy;       // force
    float rho, p;       // density, pressure
} particle_t;

particle_t *particles;
int n_particles = 0;    // number of currently active particles


/**
 * Return a random value in [a, b]
 */
float randab(float a, float b)
{
    return a + (b-a)*rand() / (float)(RAND_MAX);
}

/**
 * Set initial position of particle `*p` to (x, y); initialize all
 * other attributes to default values (zeros).
 */
void init_particle( particle_t *p, float x, float y )
{
    p->x = x;
    p->y = y;
    p->vx = p->vy = 0.0;
    p->fx = p->fy = 0.0;
    p->rho = 0.0;
    p->p = 0.0;
}

/**
 * Return nonzero iff (x, y) is within the frame
 */
int is_in_domain( float x, float y )
{
    return ((x < VIEW_WIDTH - EPS) &&
            (x > EPS) &&
            (y < VIEW_HEIGHT - EPS) &&
            (y > EPS));
}

/**
 * Initialize the SPH model with `n` particles. The caller is
 * responsible for allocating the `particles[]` array of size
 * `MAX_PARTICLES`.
 *
 * DO NOT parallelize this function, since it calls rand() which is
 * not thread-safe.
 *
 * For MPI and OpenMP: only the master must initialize the domain;
 *
 * For CUDA: the CPU must initialize the domain.
 */
void init_sph( int n )
{
    n_particles = 0;
    //printf("Initializing with %d particles\n", n);

    for (float y = EPS; y < VIEW_HEIGHT - EPS; y += H) {
        for (float x = EPS; x <= VIEW_WIDTH * 0.8f; x += H) {
            if (n_particles < n) {
                float jitter = rand() / (float)RAND_MAX;
                init_particle(particles + n_particles, x+jitter, y);
                n_particles++;
            } else {
                return;
            }
        }
    }
    assert(n_particles == n);
}

/**
 ** You may parallelize the following four functions
 **/

__device__ void compute_density_pressure( particle_t* d_particles, int * d_n_particles, int index_particle )
{
    const float HSQ = H * H;    // radius^2 for optimization

    /* Smoothing kernels defined in Muller and their gradients adapted
       to 2D per "SPH Based Shallow Water Simulation" by Solenthaler
       et al. */
    const float POLY6 = 4.0 / (M_PI * pow(H, 8));

    particle_t *pi = &d_particles[index_particle];
    pi->rho = 0.0;
    for (int j=0; j<*d_n_particles; j++) {
        const particle_t *pj = &d_particles[j];

        const float dx = pj->x - pi->x;
        const float dy = pj->y - pi->y;
        const float d2 = dx*dx + dy*dy;

        if (d2 < HSQ) {
            pi->rho += MASS * POLY6 * pow(HSQ - d2, 3.0);
        }
    }
    pi->p = GAS_CONST * (pi->rho - REST_DENS);
}

__device__ void compute_forces( particle_t* d_particles, int * d_n_particles, int index_particle )
{
    /* Smoothing kernels defined in Muller and their gradients adapted
       to 2D per "SPH Based Shallow Water Simulation" by Solenthaler
       et al. */
    const float SPIKY_GRAD = -10.0 / (M_PI * pow(H, 5));
    const float VISC_LAP = 40.0 / (M_PI * pow(H, 5));
    const float EPS = 1e-6;

    particle_t *pi = &d_particles[index_particle];
    float fpress_x = 0.0, fpress_y = 0.0;
    float fvisc_x = 0.0, fvisc_y = 0.0;

    for (int j=0; j<*d_n_particles; j++) {
        const particle_t *pj = &d_particles[j];

        if (pi == pj)
            continue;

        const float dx = pj->x - pi->x;
        const float dy = pj->y - pi->y;
        const float dist = hypotf(dx, dy) + EPS; // avoids division by zero later on

        if (dist < H) {
            const float norm_dx = dx / dist;
            const float norm_dy = dy / dist;
            // compute pressure force contribution
            fpress_x += -norm_dx * MASS * (pi->p + pj->p) / (2 * pj->rho) * SPIKY_GRAD * pow(H - dist, 3);
            fpress_y += -norm_dy * MASS * (pi->p + pj->p) / (2 * pj->rho) * SPIKY_GRAD * pow(H - dist, 3);
            // compute viscosity force contribution
            fvisc_x += VISC * MASS * (pj->vx - pi->vx) / pj->rho * VISC_LAP * (H - dist);
            fvisc_y += VISC * MASS * (pj->vy - pi->vy) / pj->rho * VISC_LAP * (H - dist);
        }
    }
    const float fgrav_x = Gx * MASS / pi->rho;
    const float fgrav_y = Gy * MASS / pi->rho;
    pi->fx = fpress_x + fvisc_x + fgrav_x;
    pi->fy = fpress_y + fvisc_y + fgrav_y;
}

__device__ void integrate( particle_t* d_particles, int index_particle )
{
    particle_t *p = &d_particles[index_particle];
    // forward Euler integration
    p->vx += DT * p->fx / p->rho;
    p->vy += DT * p->fy / p->rho;
    p->x += DT * p->vx;
    p->y += DT * p->vy;

    // enforce boundary conditions
    if (p->x - EPS < 0.0) {
        p->vx *= BOUND_DAMPING;
        p->x = EPS;
    }
    if (p->x + EPS > VIEW_WIDTH) {
        p->vx *= BOUND_DAMPING;
        p->x = VIEW_WIDTH - EPS;
    }
    if (p->y - EPS < 0.0) {
        p->vy *= BOUND_DAMPING;
        p->y = EPS;
    }
    if (p->y + EPS > VIEW_HEIGHT) {
        p->vy *= BOUND_DAMPING;
        p->y = VIEW_HEIGHT - EPS;
    }
}

__global__ void step(particle_t * d_p, int * d_n, float *d_sums) {

    const int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < *d_n) {
        compute_density_pressure(d_p, d_n, index);
        __syncthreads();

        compute_forces(d_p, d_n, index);
        __syncthreads();

        integrate(d_p, index);
        
        /* reduction of averange velocity */
        __shared__ float temp[BLKDIM];
        const int lindex = threadIdx.x;
        const int bindex = blockIdx.x;
        int bsize = blockDim.x / 2;
        temp[lindex] = hypot(d_p[index].vx, d_p[index].vy) / *d_n;

        __syncthreads();
        while ( bsize > 0 ) {
            if ( lindex < bsize ) {
                temp[lindex] += temp[lindex + bsize];
            }
            bsize = bsize / 2;
            __syncthreads();
        }
        if ( 0 == lindex ) {
            d_sums[bindex] = temp[0];
        }
        
    }

}

#define MAX_BLOCK (MAX_PARTICLES + BLKDIM - 1)/BLKDIM

int main(int argc, char **argv)
{
    srand(1234);

    particles = (particle_t*)malloc(MAX_PARTICLES * sizeof(*particles));
    assert( particles != NULL );

    int n = DAM_PARTICLES;
    int nsteps = 50;

    if (argc > 3) {
        fprintf(stderr, "Usage: %s [nparticles [nsteps]]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if (argc > 1) {
        n = atoi(argv[1]);
    }

    if (argc > 2) {
        nsteps = atoi(argv[2]);
    }

    if (n > MAX_PARTICLES) {
        fprintf(stderr, "FATAL: the maximum number of particles is %d\n", MAX_PARTICLES);
        return EXIT_FAILURE;
    }

    particle_t *d_particles;
    int *d_n_particles;
    float h_sums[MAX_BLOCK];
    //float d_sums[(MAX_PARTICLES + BLKDIM - 1) / BLKDIM];
    //float * h_sums = (float *) malloc(MAX_PARTICLES * sizeof(float));
    float *d_sums;

    init_sph(n);
    hipMalloc((void **) &d_particles, sizeof(particle_t) * MAX_PARTICLES);
    hipMemcpy(d_particles, particles, sizeof(particle_t) * MAX_PARTICLES, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_n_particles, sizeof(int));
    hipMemcpy(d_n_particles, &n, sizeof(int), hipMemcpyHostToDevice);
    
    hipMalloc((void **) &d_sums, MAX_PARTICLES * sizeof(float));

    double loop_start = hpc_gettime();
    
    for (int s=0; s<nsteps; s++) {
        double start = hpc_gettime();
        step<<<MAX_BLOCK, BLKDIM>>>(d_particles, d_n_particles, d_sums);

        /* the average velocities MUST be computed at each step, even
        if it is not shown (to ensure constant workload per
        iteration) */
        hipMemcpy(h_sums, d_sums, sizeof(h_sums), hipMemcpyDeviceToHost);
        
        float avg = 0.0;
        
        //#pragma omp simd
        for (int i = 0; i < MAX_BLOCK; i++)
            avg += h_sums[i];
        
        double end = hpc_gettime() - start;

        if (s % PRINT_AVERANGE == 0){
            //printf("step %5d, avgV=%f, took: %fs\n", s, avg, end);
            printf("%f;", avg);
            //for (int i = 0; i < MAX_BLOCK; i++)
            //    printf("%f ", h_sums[i]);
            //printf("\n");
        }
    }

    double loop_end = hpc_gettime() - loop_start;
    printf("took: %fs\n", loop_end);

    hipFree(d_particles);
    hipFree(d_n_particles);
    free(particles);
    return EXIT_SUCCESS;
}
